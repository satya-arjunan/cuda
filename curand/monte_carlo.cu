#include "hip/hip_runtime.h"
#include <thrust/iterator/counting_iterator.h>
#include <thrust/functional.h>
#include <thrust/transform_reduce.h>
#include <thrust/random.h>
#include <hiprand/hiprand_kernel.h>
#include <boost/date_time/posix_time/posix_time.hpp>

#include <iostream>
#include <iomanip>

// we could vary M & N to find the perf sweet spot

struct estimate_pi_curand : public thrust::unary_function<unsigned int, float>
{
  __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 100000; // samples per thread

    unsigned int seed = thread_id;

    hiprandState s;

    // seed a random number generator
    hiprand_init(seed, 0, 0, &s);

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      float x = hiprand_uniform(&s);
      float y = hiprand_uniform(&s);

      // measure distance from the origin
      float dist = sqrtf(x*x + y*y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if(dist <= 1.0f)
        sum += 1.0f;
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / N;
  }
};


__host__ __device__
unsigned int hash(unsigned int a)
{
    a = (a+0x7ed55d16) + (a<<12);
    a = (a^0xc761c23c) ^ (a>>19);
    a = (a+0x165667b1) + (a<<5);
    a = (a+0xd3a2646c) ^ (a<<9);
    a = (a+0xfd7046c5) + (a<<3);
    a = (a^0xb55a4f09) ^ (a>>16);
    return a;
}


struct estimate_pi_thrust : public thrust::unary_function<unsigned int,float>
{
  __host__ __device__
  float operator()(unsigned int thread_id)
  {
    float sum = 0;
    unsigned int N = 100000; // samples per thread

    //unsigned int seed = hash(thread_id);
    unsigned int seed = thread_id;

    // seed a random number generator
    thrust::default_random_engine rng(seed);

    // create a mapping from random numbers to [0,1)
    thrust::uniform_real_distribution<float> u01(0,1);

    // take N samples in a quarter circle
    for(unsigned int i = 0; i < N; ++i)
    {
      // draw a sample from the unit square
      float x = u01(rng);
      float y = u01(rng);

      // measure distance from the origin
      float dist = sqrtf(x*x + y*y);

      // add 1.0f if (u0,u1) is inside the quarter circle
      if(dist <= 1.0f)
        sum += 1.0f;
    }

    // multiply by 4 to get the area of the whole circle
    sum *= 4.0f;

    // divide by N
    return sum / N;
  }
};

void getThrustTime(const unsigned M)
{
  boost::posix_time::ptime start(
      boost::posix_time::microsec_clock::universal_time()); 
  float estimate = thrust::transform_reduce(
        thrust::counting_iterator<int>(0),
        thrust::counting_iterator<int>(M),
        estimate_pi_thrust(),
        0.0f,
        thrust::plus<float>());
  boost::posix_time::ptime end(
      boost::posix_time::microsec_clock::universal_time());
  estimate /= M;
  std::cout << std::setprecision(7);
  std::cout << "pi is approximately ";
  std::cout << estimate << " thrust time:" << (end-start) << std::endl;
}

void getCurandTime(const unsigned M)
{
  boost::posix_time::ptime start(
      boost::posix_time::microsec_clock::universal_time()); 
  float estimate = thrust::transform_reduce(
        thrust::counting_iterator<int>(0),
        thrust::counting_iterator<int>(M),
        estimate_pi_curand(),
        0.0f,
        thrust::plus<float>());
  boost::posix_time::ptime end(
      boost::posix_time::microsec_clock::universal_time());
  estimate /= M;
  std::cout << std::setprecision(7);
  std::cout << "pi is approximately ";
  std::cout << estimate << " hiprand time:" << (end-start) << std::endl;
}

int main(void)
{
  unsigned M(3000000);
  getCurandTime(M);
  getThrustTime(M);
  return 0;
}

