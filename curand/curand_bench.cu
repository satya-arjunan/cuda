#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>

//best results: 41.3 BUPS

int main(){
  size_t n = 536870912;
  int i;
  hiprandGenerator_t gen;
  float *devData;
  float f;
  hipMalloc((void **)&devData, n * sizeof(float));
  //hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_MT19937);
  hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  clock_t start=clock();
  for(i=0;i<100;i++) hiprandGenerateUniform(gen, devData, n);
  hipDeviceSynchronize();
  printf("time1 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  hipMemcpy(&f, devData, sizeof(float),hipMemcpyDeviceToHost);
  printf("time2 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  hiprandDestroyGenerator(gen);
  hipFree(devData);
  printf("time3 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  return 0;
}
