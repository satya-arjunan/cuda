#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hiprand.h>

int main(){
  size_t n = 536870912;
  clock_t start=clock();
  int i;
  hiprandGenerator_t gen;
  float *devData;
  float f;
  hipMalloc((void **)&devData, n * sizeof(float));
  hiprandCreateGenerator(&gen,HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
  for(i=0;i<100;i++) hiprandGenerateUniform(gen, devData, n);
  printf("time1 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  hipMemcpy(&f, devData, sizeof(float),hipMemcpyDeviceToHost);
  printf("time2 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  hiprandDestroyGenerator(gen);
  hipFree(devData);
  printf("time3 = %f seconds\n",(float)(clock()-start)/CLOCKS_PER_SEC);
  return 0;
}
