#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#define DCOPY_THREADS  384
#define DCOPY_DEFLEN   10000000
#define DCOPY_ITER     10           // as in STREAM benchmark

// Macro to catch CUDA errors in CUDA runtime calls
#define CUDA_SAFE_CALL(call)                                          \
do {                                                                  \
    hipError_t err = call;                                           \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// Macro to catch CUDA errors in kernel launches
#define CHECK_LAUNCH_ERROR()                                          \
do {                                                                  \
    /* Check synchronous errors, i.e. pre-launch */                   \
    hipError_t err = hipGetLastError();                             \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString(err) );       \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
    /* Check asynchronous errors, i.e. kernel failed (ULF) */         \
    err = hipDeviceSynchronize();                                    \
    if (hipSuccess != err) {                                         \
        fprintf (stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                 __FILE__, __LINE__, hipGetErrorString( err) );      \
        exit(EXIT_FAILURE);                                           \
    }                                                                 \
} while (0)

// A routine to give access to a high precision timer on most systems.
#if defined(_WIN32)
#if !defined(WIN32_LEAN_AND_MEAN)
#define WIN32_LEAN_AND_MEAN
#endif
#include <windows.h>
double second (void)
{
    LARGE_INTEGER t;
    static double oofreq;
    static int checkedForHighResTimer;
    static BOOL hasHighResTimer;

    if (!checkedForHighResTimer) {
        hasHighResTimer = QueryPerformanceFrequency (&t);
        oofreq = 1.0 / (double)t.QuadPart;
        checkedForHighResTimer = 1;
    }
    if (hasHighResTimer) {
        QueryPerformanceCounter (&t);
        return (double)t.QuadPart * oofreq;
    } else {
        return (double)GetTickCount() * 1.0e-3;
    }
}
#elif defined(__linux__) || defined(__APPLE__)
#include <stddef.h>
#include <sys/time.h>
double second (void)
{
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return (double)tv.tv_sec + (double)tv.tv_usec * 1.0e-6;
}
#else
#error unsupported platform
#endif

__global__ void dcopy (double *src, double *dst, int len)
{
    int stride = gridDim.x * blockDim.x;
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    for (int i = tid; i < len; i += stride) {
        dst[i] = src[i];
    }
}    

struct dcopyOpts {
    int len;
};

static int processArgs (int argc, char *argv[], struct dcopyOpts *opts)
{
    int error = 0;
    memset (opts, 0, sizeof(*opts));
    while (argc) {
        if (*argv[0] == '-') {
            switch (*(argv[0]+1)) {
            case 'n':
                opts->len = atol(argv[0]+2);
                break;
            default:
                fprintf (stderr, "Unknown switch '%c%s'\n", '-', argv[0]+1);
                error++;
                break;
            }
        }
        argc--;
        argv++;
    }
    return error;
}

int main (int argc, char *argv[])
{
    double start, stop, elapsed, mintime;
    double *d_a, *d_b;
    int errors;
    struct dcopyOpts opts;

    errors = processArgs (argc, argv, &opts);
    if (errors) {
        return EXIT_FAILURE;
    }
    opts.len = (opts.len) ? opts.len : DCOPY_DEFLEN;

    /* Allocate memory on device */
    CUDA_SAFE_CALL (hipMalloc((void**)&d_a, sizeof(d_a[0]) * opts.len));
    CUDA_SAFE_CALL (hipMalloc((void**)&d_b, sizeof(d_b[0]) * opts.len));
    
    /* Initialize device memory */
    CUDA_SAFE_CALL (hipMemset(d_a, 0x00, sizeof(d_a[0]) * opts.len)); // zero
    CUDA_SAFE_CALL (hipMemset(d_b, 0xff, sizeof(d_b[0]) * opts.len)); // NaN

    /* Compute execution configuration */
    dim3 dimBlock(384);
    int threadBlocks = (opts.len + (dimBlock.x - 1)) / dimBlock.x;
    if (threadBlocks > 65520) threadBlocks = 65520;
    dim3 dimGrid(threadBlocks);
    
    printf ("dcopy: operating on vectors of %d doubles (= %.3e bytes)\n", 
            opts.len, (double)sizeof(d_a[0]) * opts.len);
    printf ("dcopy: using %d threads per block, %d blocks\n", 
            dimBlock.x, dimGrid.x);

    mintime = fabs(log(0.0));
    for (int k = 0; k < DCOPY_ITER; k++) {
        start = second();
        dcopy<<<dimGrid,dimBlock>>>(d_a, d_b, opts.len);
        CHECK_LAUNCH_ERROR();
        stop = second();
        elapsed = stop - start;
        if (elapsed < mintime) mintime = elapsed;
    }
    printf ("dcopy: mintime = %.3f msec  throughput = %.2f GB/sec\n",
            1.0e3 * mintime, (2.0e-9 * sizeof(d_a[0]) * opts.len) / mintime);

    CUDA_SAFE_CALL (hipFree(d_a));
    CUDA_SAFE_CALL (hipFree(d_b));

    return EXIT_SUCCESS;
}
