#include "hip/hip_runtime.h"
//to compile with global memory: nvcc -O3 -gencode arch=compute_52,code=sm_52 -DUSE_GLOBAL surf2Dwrite_ex.cu
//to compile with surface memory: nvcc -O3 -gencode arch=compute_52,code=sm_52 surf2Dwrite_ex.cu

#include <stdio.h>
#include <iostream>

typedef float mytype;
const int blk_dim=16;

#define my_N 1000
#define A_VAL 1
#define B_VAL 2

surface < void, 2 > a_surf;
surface < void, 2 > b_surf;
surface < void, 2 > c_surf;

void CUDA_SAFE_CALL(hipError_t call, int line) {
    switch (call) {
    case hipSuccess:
        break;
    default:
        printf("ERROR at line :%i.%d' ' %s\n",
            line, call, hipGetErrorString(call));
        exit(-1);
        break;
    }

}

#ifdef USE_GLOBAL
__global__ void mul(const mytype * __restrict__ d_a, const mytype * __restrict__ d_b, mytype * __restrict__ d_c, const int N)
#else
__global__ void mul(const int N)
#endif
{
    mytype a, b, c, temp;
    int i;

    unsigned int x = blockIdx.x * blockDim.x + (threadIdx.x);
    unsigned int y = blockIdx.y * blockDim.y + (threadIdx.y);
    if (x < N && y < N) {

        temp = 0;
        for (i = 0; i < N; i++) {
#ifdef USE_GLOBAL
            a = d_a[x*N+i];
            b = d_b[i*N+y];
#else
            surf2Dread( & a, a_surf, (x) * sizeof(mytype), i);
            surf2Dread( & b, b_surf, (i) * sizeof(mytype), y);
#endif
            temp += a * b;
        }
        c = temp;
#ifdef USE_GLOBAL
        d_c[x*N+y] = c;
#else
        // Write to output surface
        surf2Dwrite(c, c_surf, x * sizeof(mytype), y);
#endif
    }
}

int main() {
    const int N = my_N;
    mytype *a, *b, *c, *d_a, *d_b, *d_c;
    int i, j;
    clock_t t1, t2;
    hipArray * da, * db, * dc;
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc < mytype > ();

    dim3 dimBlock(blk_dim, blk_dim);
    dim3 dimGrid((N+dimBlock.x-1)/dimBlock.x, (N+dimBlock.y-1)/dimBlock.y);
    int s = N * N * sizeof(mytype);

    a = (mytype *)malloc(s);
    b = (mytype *)malloc(s);
    c = (mytype *)malloc(s);

    CUDA_SAFE_CALL(hipMalloc(&d_a, s), __LINE__);
    CUDA_SAFE_CALL(hipMalloc(&d_b, s), __LINE__);
    CUDA_SAFE_CALL(hipMalloc(&d_c, s), __LINE__);

    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            a[i*N+j] = A_VAL;

    for (i = 0; i < N; i++)
        for (j = 0; j < N; j++)
            b[i*N+j] = B_VAL;

    CUDA_SAFE_CALL(hipMallocArray( & da, & channelDesc, N, N, hipArraySurfaceLoadStore), __LINE__);
    CUDA_SAFE_CALL(hipMallocArray( & db, & channelDesc, N, N, hipArraySurfaceLoadStore), __LINE__);
    CUDA_SAFE_CALL(hipMallocArray( & dc, & channelDesc, N, N, hipArraySurfaceLoadStore), __LINE__);


    CUDA_SAFE_CALL(hipMemcpyToArray(da, 0, 0, a, s, hipMemcpyHostToDevice), __LINE__);
    CUDA_SAFE_CALL(hipMemcpyToArray(db, 0, 0, b, s, hipMemcpyHostToDevice), __LINE__);

    CUDA_SAFE_CALL(cudaBindSurfaceToArray(a_surf, da), __LINE__);
    CUDA_SAFE_CALL(cudaBindSurfaceToArray(b_surf, db), __LINE__);
    CUDA_SAFE_CALL(cudaBindSurfaceToArray(c_surf, dc), __LINE__);

#ifdef USE_GLOBAL
    CUDA_SAFE_CALL(hipMemcpy(d_a, a, s, hipMemcpyHostToDevice), __LINE__);
    CUDA_SAFE_CALL(hipMemcpy(d_b, b, s, hipMemcpyHostToDevice), __LINE__);
#endif
    t1 = clock();
#ifdef USE_GLOBAL
    mul <<<dimGrid, dimBlock>>> (d_a, d_b, d_c, N);
#else
    mul <<<dimGrid, dimBlock>>> (N);
#endif
    hipDeviceSynchronize();
    t2 = clock();

    CUDA_SAFE_CALL(hipMemcpyFromArray(c, dc, 0, 0, s, hipMemcpyDeviceToHost), __LINE__);
#ifdef USE_GLOBAL
    CUDA_SAFE_CALL(hipMemcpy(c, d_c, s, hipMemcpyDeviceToHost), __LINE__);
#endif

    double t3 = (double) t2 - (double) t1;
    t3 = t3 / CLOCKS_PER_SEC;

    printf("\n CUDA time :%lf\n", t3);
    for (i=0; i < N*N; i++)
      if(c[i] != A_VAL*B_VAL*N) {std::cout << "mismatch at: " << i << ", was: " << c[i] << " should be: " << A_VAL*B_VAL*N << std::endl;  return 1;}

    CUDA_SAFE_CALL(hipFreeArray(da), __LINE__);
    CUDA_SAFE_CALL(hipFreeArray(db), __LINE__);
    CUDA_SAFE_CALL(hipFreeArray(dc), __LINE__);
    std::cout << "Success!"  << std::endl;
    return 0;
}
